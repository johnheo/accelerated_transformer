
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define n 1024
#define T 196
#define D 384
#define H 12
__global__ void matrixmult_Q_K(int *a, int *b, int *c){
	int X = blockIdx.x*blockDim.x + threadIdx.x;
	int Y = blockIdx.y*blockDim.y + threadIdx.y;
	int local_c = 0;
	if(X < T && Y < T){
		for (int i = 0; i< D/H; i++)
			local_c += a[X*D/H + i] * b[i*T + Y];
		c[X*T + Y] = local_c;  
	}
}
__global__ void matrixmult_QK_V(int *a, int *b, int *c){
	int X = blockIdx.x*blockDim.x + threadIdx.x;
	int Y = blockIdx.y*blockDim.y + threadIdx.y;
	int local_c = 0;
	if(X < T && Y < D/H){
		for (int i = 0; i< T; i++)
			local_c += a[X*T + i] * b[i*D/H + Y];
		c[X*D/H + Y] = local_c;  
	}
}



int main(){	
	float ms; // elapsed time in milliseconds

    hipEvent_t startEvent,stopEvent;
	hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    int i;
    int *Q = (int*)malloc(sizeof(int)*T*D/H);         
	int *K = (int*)malloc(sizeof(int)*D/H*T);         
    int *QK= (int*)malloc(sizeof(int)*T*T);           
    int *QKV=(int*)malloc(sizeof(int)*T*D/H);        
    int *V = (int*)malloc(sizeof(int)*T*D/H);           	

	for(i=0; i<T*D/H; i++){
		Q[i]=1;
		K[i]=2;
		V[i]=1;
	}
	
	struct timespec start, stop; 
	double time;
	if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}

	hipEventRecord(startEvent,0);


	int *gpu_Q, *gpu_K, *gpu_QK, *gpu_V, *gpu_QKV;
	hipMalloc((void**)&gpu_Q, sizeof(int)*T*D/H); 
	hipMalloc((void**)&gpu_K, sizeof(int)*D/H*T);
	hipMalloc((void**)&gpu_QK, sizeof(int)*T*T);
	hipMalloc((void**)&gpu_V, sizeof(int)*T*D/H);
	hipMalloc((void**)&gpu_QKV, sizeof(int)*T*D/H);
		

  
  
	hipMemcpy(gpu_Q, Q, sizeof(int)*T*D/H, hipMemcpyHostToDevice);
	hipMemcpy(gpu_K, K, sizeof(int)*D/H*T, hipMemcpyHostToDevice);
	hipMemcpy(gpu_V, V, sizeof(int)*D/H*T, hipMemcpyHostToDevice);
	
	dim3 dimGrid_1(T/16,T/16);
	dim3 dimBlock_1(16,16);
	dim3 dimGrid_2(T/16,D/H/16);
	dim3 dimBlock_2(16,16);
	
	matrixmult_Q_K<<<dimGrid_1, dimBlock_1>>>(gpu_Q, gpu_K, gpu_QK);
	
	matrixmult_QK_V<<<dimGrid_2, dimBlock_2>>>(gpu_QK, gpu_V, gpu_QKV);

	
	hipEventRecord(stopEvent, 0) ;
	hipEventSynchronize(stopEvent) ;
	hipEventElapsedTime(&ms, startEvent, stopEvent) ;
	printf("Standard Time execute using cuda api(ms): %f   \n", ms);
	
	if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	  
	time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
	//printf("standard attention time is %f ns\n", time*1e9);	 
	
	hipMemcpy(QK, gpu_QK, sizeof(int)*T*T, hipMemcpyDeviceToHost);
	hipMemcpy(QKV, gpu_QKV, sizeof(int)*T*D/H, hipMemcpyDeviceToHost);


	/*
	printf("QK[100][100] = %f\n",QK[100*T+100]);
	printf("should equal 2*D/H = 128\n");
	printf("QKV[100][30] = %f\n",QKV[100*D/H+30]);
	printf("should equal 196* 128 = 25088");
	*/
	return 0;
}	
