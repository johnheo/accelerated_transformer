
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define n 1024
#define T 196
#define D 384
#define H 12
__global__ void matrixmult_Q_K(int *a, int *b, int *c){
	int X = blockIdx.x*blockDim.x + threadIdx.x;
	int Y = blockIdx.y*blockDim.y + threadIdx.y;
	int local_c = 0;
	if(X < T && Y < T){
		for (int i = 0; i< D/H; i++)
			local_c += a[X*D/H + i] * b[i*T + Y];
		c[X*T + Y] = local_c;  
	}
}
__global__ void matrixmult_QK_V(int *a, int *b, int *c){
	int X = blockIdx.x*blockDim.x + threadIdx.x;
	int Y = blockIdx.y*blockDim.y + threadIdx.y;
	int local_c = 0;
	if(X < T && Y < D/H){
		for (int i = 0; i< T; i++)
			local_c += a[X*T + i] * b[i*D/H + Y];
		c[X*D/H + Y] = local_c;  
	}
}



int main(){	
	
    int i;
    int *Q = (int*)malloc(sizeof(int)*T*D/H);          
	int *K = (int*)malloc(sizeof(int)*D/H*T);          
    int *QK= (int*)malloc(sizeof(int)*T*T);           	
    int *QKV=(int*)malloc(sizeof(int)*T*D/H);           	

    int *V = (int*)malloc(sizeof(int)*T*D/H);           	

	for(i=0; i<T*D/H; i++){
		Q[i]=1;
		K[i]=2;
		V[i]=1;
	}
	
	int *gpu_Q, *gpu_K, *gpu_QK, *gpu_V, *gpu_QKV;
	hipMalloc((void**)&gpu_Q, sizeof(int)*T*D/H); 
	hipMalloc((void**)&gpu_K, sizeof(int)*D/H*T);
	hipMalloc((void**)&gpu_QK, sizeof(int)*T*T);
	hipMalloc((void**)&gpu_V, sizeof(int)*T*D/H);
	hipMalloc((void**)&gpu_QKV, sizeof(int)*T*D/H);
		
	struct timespec start, stop; 
	double time;
  
  
	hipMemcpy(gpu_Q, Q, sizeof(int)*T*D/H, hipMemcpyHostToDevice);
	hipMemcpy(gpu_K, K, sizeof(int)*D/H*T, hipMemcpyHostToDevice);
	hipMemcpy(gpu_V, V, sizeof(int)*D/H*T, hipMemcpyHostToDevice);
	
	dim3 dimGrid_1(16,16);
	dim3 dimBlock_1(16,16);
	dim3 dimGrid_2(16,4);
	dim3 dimBlock_2(16,16);
	
	if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}
	matrixmult_Q_K<<<dimGrid_1, dimBlock_1>>>(gpu_Q, gpu_K, gpu_QK);
	
	matrixmult_QK_V<<<dimGrid_2, dimBlock_2>>>(gpu_QK, gpu_V, gpu_QKV);				
	
	if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	  
	time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec);
	printf("Standard attention time is %d ns || %.6f s \n", time, time/1e9);	 
	
	hipMemcpy(QK, gpu_QK, sizeof(int)*T*T, hipMemcpyDeviceToHost);
	hipMemcpy(QKV, gpu_QKV, sizeof(int)*T*D/H, hipMemcpyDeviceToHost);


	/*
	printf("QK[100][100] = %d\n",QK[100*T+100]);
	printf("should equal 2*D/H = 128\n");
	printf("QKV[100][30] = %d\n",QKV[100*D/H+30]);
	printf("should equal 196* 128 = 25088/n");
*/
	return 0;
}	
