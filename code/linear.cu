
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define n 1024
#define T 196
#define D 384
#define H 12
__global__ void matrixmult_K_V(int *a, int *b, int *c){
	int X = blockIdx.x*blockDim.x + threadIdx.x;
	int Y = blockIdx.y*blockDim.y + threadIdx.y;
	int local_c = 0;
	if(X < D/H && Y < D/H){
		for (int i = 0; i< T; i++)
			local_c += a[X*T + i] * b[i*D/H + Y];
		c[X*D/H + Y] = local_c;  
	}
}
__global__ void matrixmult_Q_KV(int *a, int *b, int *c){
	int X = blockIdx.x*blockDim.x + threadIdx.x;
	int Y = blockIdx.y*blockDim.y + threadIdx.y;
	int local_c = 0;
	if(X < T && Y < D/H){
		for (int i = 0; i< D/H; i++)
			local_c += a[X*D/H + i] * b[i*D/H + Y];
		c[X*D/H + Y] = local_c;  
	}
}



int main(){	
	
    int i;
    int *Q = (int*)malloc(sizeof(int)*T*D/H);          
	int *K = (int*)malloc(sizeof(int)*D/H*T);          
    int *KV= (int*)malloc(sizeof(int)*D/H*D/H);           	
    int *QKV=(int*)malloc(sizeof(int)*T*D/H);           	
    int *V = (int*)malloc(sizeof(int)*T*D/H);           	

	for(i=0; i<T*D/H; i++){
		Q[i]=1;
		K[i]=2;
		V[i]=1;
	}
	
	int *gpu_Q, *gpu_K, *gpu_KV, *gpu_V, *gpu_QKV;
	hipMalloc((void**)&gpu_Q, sizeof(int)*T*D/H); 
	hipMalloc((void**)&gpu_K, sizeof(int)*D/H*T);
	hipMalloc((void**)&gpu_KV, sizeof(int)*D/H*D/H);
	hipMalloc((void**)&gpu_V, sizeof(int)*T*D/H);
	hipMalloc((void**)&gpu_QKV, sizeof(int)*T*D/H);
		
	struct timespec start, stop; 
	double time;
  
  
	hipMemcpy(gpu_Q, Q, sizeof(int)*T*D/H, hipMemcpyHostToDevice);
	hipMemcpy(gpu_K, K, sizeof(int)*D/H*T, hipMemcpyHostToDevice);
	hipMemcpy(gpu_V, V, sizeof(int)*D/H*T, hipMemcpyHostToDevice);
	
	dim3 dimGrid_1(4,4);
	dim3 dimBlock_1(16,16);
	dim3 dimGrid_2(16,4);
	dim3 dimBlock_2(16,16);
	
	if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}
	matrixmult_K_V<<<dimGrid_1, dimBlock_1>>>(gpu_K, gpu_V, gpu_KV);
	
	matrixmult_Q_KV<<<dimGrid_2, dimBlock_2>>>(gpu_Q, gpu_KV, gpu_QKV);				
	
	if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	  
	time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
	printf("Linear attention time is %f ns\n", time*1e9);	 
	
	hipMemcpy(KV, gpu_KV, sizeof(int)*D/H*D/H, hipMemcpyDeviceToHost);
	hipMemcpy(QKV, gpu_QKV, sizeof(int)*T*D/H, hipMemcpyDeviceToHost);


/*
	printf("QK[50][50] = %d\n",KV[50*D/H+50]);
	printf("should equal 2*T = 392\n");
	printf("QKV[100][30] = %d\n",QKV[100*D/H+30]);
	printf("should equal 392* 64 = 25088");
*/
	return 0;
}	
